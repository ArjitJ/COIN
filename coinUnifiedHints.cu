#include "hip/hip_runtime.h"
#include <cstdio> 
#include <cstdlib> 
#include <cmath>
#include <cstring>
#include <string>
#include <cassert>
#include <iostream>
#include <fstream>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include ""

using namespace std;
__global__ void copyBias(float *O, float *Z, int N, int M) {
    int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
    if(tid<N){
        O[tid] = Z[tid%M];
    }
}

__global__ void sineActivation(float *O, float *Z, int N, float weight=30.0) {
    int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
    if(tid<N){
        O[tid] = sin(weight*Z[tid]);
    }
}
void readIntoArray(float* arr, ifstream* inFile, int SIZE){
	if (inFile->is_open())  
    {
        for (int i = 0; i < SIZE; i++) 
        {
            *inFile >> arr[i];
        }
        inFile->close();
    }
}
void fillCoordinateMatrix(float* X, int STARTX, int STARTY, int ENDX, int ENDY, int RESX, int RESY, int HEIGHT, int WIDTH){
    float start_x = STARTX/(HEIGHT-1.0);
    start_x -= 0.5;
    start_x *= 2.0;
    float start_y = STARTY/(HEIGHT-1.0);
    start_y -= 0.5;
    start_y *= 2.0;
    float diff_x = 2*((ENDX-STARTX)/(HEIGHT-1.0))/RESX;
    float diff_y = 2*((ENDY-STARTY)/(HEIGHT-1.0))/RESY;
    int idx=0;
    float tmp = start_y;
    for(int i=0;i<RESX;i++){
        for(int j=0;j<RESY;j++){
            X[idx++] = start_x;
            X[idx++] = tmp;
            tmp += diff_y;
        }
        start_x += diff_x;
        tmp = start_y;
    }
}
int main(int argc, char* argv[]){

    int INP_DIM = 2;
    int OUT_DIM = 3;

    // ArgParse
    int NUM_LAYERS, DIM, HEIGHT, WIDTH, RESX, RESY, STARTX, STARTY, ENDX, ENDY;
    NUM_LAYERS = atoi(argv[1]);
    DIM = atoi(argv[2]);
    HEIGHT = atoi(argv[3]);
    WIDTH = atoi(argv[4]);
    RESX = atoi(argv[5]);
    RESY = atoi(argv[6]);
    STARTX = atoi(argv[7]);
    STARTY = atoi(argv[8]);
    ENDX = atoi(argv[9]);
    ENDY = atoi(argv[10]);
    
    ifstream inFile;
	float* W;
	float* B;
	float* Z;
	float* X;
    
    int weightSize = DIM*DIM;
    int biasSize = DIM;
    int COORDS = RESX*RESY;
    int outputSize = COORDS*DIM;
    float alpha = 1.0f;
    float beta = 1.0f;
    
    int idx = 0;
    int NUM_THREADS=1024;
    int NUM_BLOCKS;
	
	
    float time;
    hipEvent_t start, stop;	
    
    hipblasHandle_t handle;
    hipblasCreate(&handle);
	
    int id = hipGetDevice(&id);	

    hipMallocManaged(&Z, outputSize*sizeof(float));
    hipMallocManaged(&W, weightSize*sizeof(float));
    hipMallocManaged(&B, biasSize*sizeof(float));
    hipMallocManaged(&X, COORDS*DIM*sizeof(float));
    
    hipMemPrefetchAsync(Z, outputSize*sizeof(float), hipCpuDeviceId);
//	hipMemAdvise(W, weightSize*sizeof(float), hipMemAdviseSetPreferredLocation, id);
    hipMemAdvise(B, biasSize*sizeof(float), hipMemAdviseSetPreferredLocation, hipCpuDeviceId);   
//	hipMemAdvise(X, COORDS*DIM*sizeof(float), hipMemAdviseSetPreferredLocation, hipCpuDeviceId);   
// 	hipMemPrefetchAsync(Z, outputSize*sizeof(float), id);
    hipMemAdvise(W, weightSize*sizeof(float), hipMemAdviseSetReadMostly, id);

    fillCoordinateMatrix(X, STARTX, STARTY, ENDX, ENDY, RESX, RESY, HEIGHT, WIDTH);
	hipMemPrefetchAsync(X, COORDS*DIM*sizeof(float), id);
    hipMemAdvise(X, COORDS*DIM*sizeof(float), hipMemAdviseSetPreferredLocation, id);   
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
    NUM_BLOCKS=ceil((float)(COORDS*DIM)/NUM_THREADS);
    for(int layer=0;layer<NUM_LAYERS;layer++){
        string weightsfileName = "weightsT/net."+to_string(layer)+".linear.weight";
        string biasfileName = "weightsT/net."+to_string(layer)+".linear.bias";
        inFile.open(weightsfileName.c_str());
        if(layer == 0){
            readIntoArray(W, &inFile, DIM*INP_DIM);
        }
        else{
            readIntoArray(W, &inFile, weightSize);
        }
        hipMemPrefetchAsync(W, weightSize*sizeof(float), id);
  
        inFile.open(biasfileName.c_str());
        readIntoArray(B, &inFile, biasSize);
        idx=0;
        /*
        for(int j=0;j<COORDS;j++){
            for(int i=0;i<biasSize;i++){
        		Z[idx++] = B[i];
        	}
		}
        */
        copyBias<<<NUM_BLOCKS, NUM_THREADS>>>(Z, B, COORDS*biasSize, biasSize);
        hipDeviceSynchronize();
        hipMemPrefetchAsync(Z, outputSize*sizeof(float), id);
        if(layer == 0){
            hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, DIM, COORDS, INP_DIM, &alpha, W, DIM, X, INP_DIM,
                    &beta, Z, DIM);
        }
        else{
            hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, DIM, COORDS, DIM, &alpha, W, DIM, X, DIM,
                    &beta, Z, DIM);
        }
        hipDeviceSynchronize();
        sineActivation<<<NUM_BLOCKS, NUM_THREADS>>>(X, Z, COORDS*DIM);
        hipDeviceSynchronize();
        hipMemPrefetchAsync(Z, outputSize*sizeof(float), hipCpuDeviceId);
        hipMemPrefetchAsync(W, weightSize*sizeof(float), hipCpuDeviceId);
    }
    hipMemAdvise(X, COORDS*DIM*sizeof(float), hipMemAdviseSetReadMostly, id);
    
    string weightsfileName = "weightsT/last_layer.linear.weight";
    string biasfileName = "weightsT/last_layer.linear.bias";
    inFile.open(weightsfileName.c_str());
    readIntoArray(W, &inFile, DIM*OUT_DIM);
    hipMemPrefetchAsync(W, weightSize*sizeof(float), id);

    inFile.open(biasfileName.c_str());
    readIntoArray(B, &inFile, OUT_DIM);
    idx=0;

    for(int j=0;j<COORDS;j++){
        for(int i=0;i<biasSize;i++){
            Z[idx++] = B[i];
        }
    }
    hipMemPrefetchAsync(Z, outputSize*sizeof(float), id);
    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, OUT_DIM, COORDS, DIM, &alpha, W, OUT_DIM, X, DIM,
            &beta, Z, OUT_DIM);
    hipDeviceSynchronize();

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    cout<<"Time Taken: "<<time/1000<<endl;
    /*
    idx = 0;
    for(int i=0;i<COORDS;i++){
    	for(int j=0;j<OUT_DIM;j++){
    		cout<<Z[idx++]<<endl;
    	}
    }
    */
    hipFree(W);
    hipFree(Z);
    hipFree(B);
    hipFree(X);
}
