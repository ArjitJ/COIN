#include <cstdio> 
#include <cstdlib> 
#include <cmath>
#include <cstring>
#include <string>
#include <cassert>
#include <iostream>
#include <fstream>
#include <hip/hip_runtime.h>
#include <hipblas.h>


using namespace std;
__global__ void sineActivation(float *O, float *Z, int N, float weight=30.0) {
    int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
    if(tid<N){
        O[tid] = sin(weight*Z[tid]);
    }
}
void readIntoArray(float* arr, ifstream* inFile, int SIZE){
	if (inFile->is_open())  
    {
        for (int i = 0; i < SIZE; i++) 
        {
            *inFile >> arr[i];
        }
        inFile->close();
    }
}
void fillCoordinateMatrix(float* X, int STARTX, int STARTY, int ENDX, int ENDY, int RESX, int RESY, int HEIGHT, int WIDTH){
    float start_x = STARTX/(HEIGHT-1.0);
    start_x -= 0.5;
    start_x *= 2.0;
    float start_y = STARTY/(HEIGHT-1.0);
    start_y -= 0.5;
    start_y *= 2.0;
    float diff_x = 2*((ENDX-STARTX)/(HEIGHT-1.0))/RESX;
    float diff_y = 2*((ENDY-STARTY)/(HEIGHT-1.0))/RESY;
    int idx=0;
    float tmp = start_y;
    for(int i=0;i<RESX;i++){
        for(int j=0;j<RESY;j++){
            X[idx++] = start_x;
            X[idx++] = tmp;
            tmp += diff_y;
        }
        start_x += diff_x;
        tmp = start_y;
    }
}
int main(int argc, char* argv[]){

    int INP_DIM = 2;
    int OUT_DIM = 3;

    // ArgParse
    int NUM_LAYERS, DIM, HEIGHT, WIDTH, RESX, RESY, STARTX, STARTY, ENDX, ENDY;
    NUM_LAYERS = atoi(argv[1]);
    DIM = atoi(argv[2]);
    HEIGHT = atoi(argv[3]);
    WIDTH = atoi(argv[4]);
    RESX = atoi(argv[5]);
    RESY = atoi(argv[6]);
    STARTX = atoi(argv[7]);
    STARTY = atoi(argv[8]);
    ENDX = atoi(argv[9]);
    ENDY = atoi(argv[10]);
    
    ifstream inFile;
	float* W;
	float* B;
	float* Z;
	float* X;
    
    int weightSize = DIM*DIM;
    int biasSize = DIM;
    int COORDS = RESX*RESY;
    int outputSize = COORDS*DIM;
    float alpha = 1.0f;
    float beta = 1.0f;
    
    int idx = 0;
    int NUM_THREADS=1024;
    int NUM_BLOCKS;
	
	
    float time;
    hipEvent_t start, stop;	
    
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    hipMallocManaged(&Z, outputSize*sizeof(float));
    hipMallocManaged(&W, weightSize*sizeof(float));
    hipMallocManaged(&B, biasSize*sizeof(float));
    hipMallocManaged(&X, COORDS*DIM*sizeof(float));


    fillCoordinateMatrix(X, STARTX, STARTY, ENDX, ENDY, RESX, RESY, HEIGHT, WIDTH);
    
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

    for(int layer=0;layer<NUM_LAYERS;layer++){
    
        string weightsfileName = "weightsT/net."+to_string(layer)+".linear.weight";
        string biasfileName = "weightsT/net."+to_string(layer)+".linear.bias";
        inFile.open(weightsfileName.c_str());
        if(layer == 0){
            readIntoArray(W, &inFile, DIM*INP_DIM);
        }
        else{
            readIntoArray(W, &inFile, weightSize);
        }
        inFile.open(biasfileName.c_str());
        readIntoArray(B, &inFile, biasSize);

        idx=0;
        for(int j=0;j<COORDS;j++){
            for(int i=0;i<biasSize;i++){
        		Z[idx++] = B[i];
        	}
        }

        if(layer == 0){
            hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, DIM, COORDS, INP_DIM, &alpha, W, DIM, X, INP_DIM,
                    &beta, Z, DIM);
        }
        else{
            hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, DIM, COORDS, DIM, &alpha, W, DIM, X, DIM,
                    &beta, Z, DIM);
        }

        hipDeviceSynchronize();
        NUM_BLOCKS=ceil((float)(COORDS*DIM)/NUM_THREADS);
        sineActivation<<<NUM_BLOCKS, NUM_THREADS>>>(X, Z, COORDS*DIM);
        hipDeviceSynchronize();
    }

    string weightsfileName = "weightsT/last_layer.linear.weight";
    string biasfileName = "weightsT/last_layer.linear.bias";
    inFile.open(weightsfileName.c_str());
    readIntoArray(W, &inFile, DIM*OUT_DIM);
    inFile.open(biasfileName.c_str());
    readIntoArray(B, &inFile, OUT_DIM);
    idx=0;

    for(int j=0;j<COORDS;j++){
        for(int i=0;i<biasSize;i++){
            Z[idx++] = B[i];
        }
    }

    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, OUT_DIM, COORDS, DIM, &alpha, W, OUT_DIM, X, DIM,
            &beta, Z, OUT_DIM);
    hipDeviceSynchronize();

    idx = 0;
    for(int i=0;i<COORDS;i++){
    	for(int j=0;j<OUT_DIM;j++){
    		cout<<Z[idx++]<<endl;
    	}
    }
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);
	cout<<"Time Taken: "<<time/1000<<endl;

    hipFree(W);
    hipFree(Z);
    hipFree(B);
    hipFree(X);
}
