#include <cstdio> 
#include <cstdlib> 
#include <cmath>
#include <cstring>
#include <string>
#include <cassert>
#include <iostream>
#include <fstream>
#include <hip/hip_runtime.h>
#include <hipblas.h>


using namespace std;
__global__ void MatrixMultiply(int M, int N, int K, float* A, int LDA, float* B, int LDB, float*C, int LDC) {
  __shared__ float ABlock[32*32];
  __shared__ float BBlock[32*32];
  int b = 32;
  int tx = threadIdx.x;
  int ty = threadIdx.y;
  int bx = blockIdx.x;
  int by = blockIdx.y;
  int i, j, X_MAX, Y_MAX, K_MAX;
  i = bx*b;
  j = by*b;
  int xoffset = i+tx;
  int yoffset = j+ty;
  int xbase = tx*b;
  int xbaselong = tx*LDB;
  int fixedbaseXA = xoffset*LDA;
  int fixedbaseXC = xoffset*LDC;
  int increment = b*LDB;
  int kbase, fixedbaseKA, fixedbaseKB;
  int cell = xbase + ty;
  X_MAX = min(M, i+b);
  Y_MAX = min(N, j+b);
  int outOfBoundX = xoffset>=X_MAX;
  int outOfBoundY = yoffset>=Y_MAX;
  if(outOfBoundX && outOfBoundY){
    return;
  }
  float temp = 0;
  fixedbaseKA = fixedbaseXA + ty;
  fixedbaseKB = xbaselong+yoffset;
  for(int k=0;k<K;k+=b){
    K_MAX = min(K, k+b)-k;
    if(!outOfBoundX && ty<K_MAX){
      ABlock[cell] = A[fixedbaseKA];
    }
    if(!outOfBoundY && tx<K_MAX){
      BBlock[cell] = B[fixedbaseKB];
    }
    __syncthreads();
    if(!outOfBoundX && !outOfBoundY){
      kbase = ty;
      for(int koffset=0;koffset<K_MAX;koffset++){
        temp += ABlock[xbase+koffset]*BBlock[kbase];
        kbase += b;
      }
    }
    __syncthreads(); 
    fixedbaseKB += increment;
    fixedbaseKA += b;
  }
  if(!outOfBoundX && !outOfBoundY){
    C[fixedbaseXC+yoffset] += temp;
  }
}
__global__ void sineActivation(float *O, float *Z, int N, float weight=30.0) {
    int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
    if(tid<N){
        O[tid] = sin(weight*Z[tid]);
    }
}
void readIntoArray(float* arr, ifstream* inFile, int SIZE){
	if (inFile->is_open())  
    {   

        for (int i = 0; i < SIZE; i++) 
        {
            *inFile >> arr[i];
        }
        inFile->close();
    }
}
void fillCoordinateMatrix(float* X, int STARTX, int STARTY, int ENDX, int ENDY, int RESX, int RESY, int HEIGHT, int WIDTH){
    float start_x = STARTX/(HEIGHT-1.0);
    start_x -= 0.5;
    start_x *= 2.0;
    float start_y = STARTY/(HEIGHT-1.0);
    start_y -= 0.5;
    start_y *= 2.0;
    float diff_x = 2*((ENDX-STARTX)/(HEIGHT-1.0))/RESX;
    float diff_y = 2*((ENDY-STARTY)/(HEIGHT-1.0))/RESY;
    int idx=0;
    float tmp = start_y;
    for(int i=0;i<RESX;i++){
        for(int j=0;j<RESY;j++){
            X[idx++] = start_x;
            X[idx++] = tmp;
            tmp += diff_y;
        }
        start_x += diff_x;
        tmp = start_y;
    }
}
int main(int argc, char* argv[]){

    int INP_DIM = 2;
    int OUT_DIM = 3;

    // ArgParse
    int NUM_LAYERS, DIM, HEIGHT, WIDTH, RESX, RESY, STARTX, STARTY, ENDX, ENDY;
    NUM_LAYERS = atoi(argv[1]);
    DIM = atoi(argv[2]);
    HEIGHT = atoi(argv[3]);
    WIDTH = atoi(argv[4]);
    RESX = atoi(argv[5]);
    RESY = atoi(argv[6]);
    STARTX = atoi(argv[7]);
    STARTY = atoi(argv[8]);
    ENDX = atoi(argv[9]);
    ENDY = atoi(argv[10]);
    
    ifstream inFile;
	  float* cpuW;
    float* cpuB;
    float* cpuZ;
    float* cpuX;
    float* gpuW;
    float* gpuB;
    float* gpuZ;
    float* gpuX;
    
    int weightSize = DIM*DIM;
    int biasSize = DIM;
    int COORDS = RESX*RESY;
    int outputSize = COORDS*DIM;
    
    int idx = 0;
    int NUM_THREADS=1024;
    int NUM_BLOCKS;
	  int b=32;
	  int MULTHREADS = 32;
    int MULBLOCKSX;
    int MULBLOCKSY;
    dim3 threads(MULTHREADS, MULTHREADS);
      
    float time;
    hipEvent_t start, stop;	
    
    cpuZ = new float[outputSize];
    cpuW = new float[weightSize];
    cpuB = new float[biasSize];
    cpuX = new float[COORDS*DIM];
  
    hipMalloc(&gpuZ, outputSize*sizeof(float));
    hipMalloc(&gpuW, weightSize*sizeof(float));
    hipMalloc(&gpuB, biasSize*sizeof(float));
    hipMalloc(&gpuX, COORDS*DIM*sizeof(float));

    fillCoordinateMatrix(cpuX, STARTX, STARTY, ENDX, ENDY, RESX, RESY, HEIGHT, WIDTH);
    hipMemcpy(gpuX, cpuX, COORDS*DIM*sizeof(float), hipMemcpyHostToDevice);
    
  	hipEventCreate(&start);
  	hipEventCreate(&stop);
  	hipEventRecord(start, 0);

    for(int layer=0;layer<NUM_LAYERS;layer++){
        string weightsfileName = "weightsT/net."+to_string(layer)+".linear.weight";
        string biasfileName = "weightsT/net."+to_string(layer)+".linear.bias";
        inFile.open(weightsfileName.c_str());
        if(layer == 0){
            readIntoArray(cpuW, &inFile, DIM*INP_DIM);
        }
        else{
            readIntoArray(cpuW, &inFile, weightSize);
        }
        inFile.open(biasfileName.c_str());
        readIntoArray(cpuB, &inFile, biasSize);
        hipMemcpy(gpuW, cpuW, weightSize*sizeof(float), hipMemcpyHostToDevice);
        idx=0;
        for(int j=0;j<COORDS;j++){
            for(int i=0;i<biasSize;i++){
        		cpuZ[idx++] = cpuB[i];
        	}
        }
        hipMemcpy(gpuZ, cpuZ, outputSize*sizeof(float), hipMemcpyHostToDevice);

        MULBLOCKSX = ceil((float)COORDS/b);
        MULBLOCKSY = ceil((float)DIM/b);
        dim3 blocks(MULBLOCKSX, MULBLOCKSY);
        if(layer == 0){
            MatrixMultiply<<<blocks,threads>>>(COORDS, DIM, INP_DIM, gpuX, INP_DIM, gpuW, DIM, gpuZ, DIM);
        }
        else{
            MatrixMultiply<<<blocks,threads>>>(COORDS, DIM, DIM, gpuX, DIM, gpuW, DIM, gpuZ, DIM);
        }
        hipDeviceSynchronize();
        NUM_BLOCKS=ceil((float)(COORDS*DIM)/NUM_THREADS);
        sineActivation<<<NUM_BLOCKS, NUM_THREADS>>>(gpuX, gpuZ, COORDS*DIM);
        hipDeviceSynchronize();
    }
    string weightsfileName = "weightsT/last_layer.linear.weight";
    string biasfileName = "weightsT/last_layer.linear.bias";
    inFile.open(weightsfileName.c_str());
    readIntoArray(cpuW, &inFile, DIM*OUT_DIM);
    inFile.open(biasfileName.c_str());
    readIntoArray(cpuB, &inFile, OUT_DIM);
    idx=0;
    hipMemcpy(gpuW, cpuW, weightSize*sizeof(float), hipMemcpyHostToDevice);

    for(int j=0;j<COORDS;j++){
        for(int i=0;i<biasSize;i++){
            cpuZ[idx++] = cpuB[i];
        }
    }
    hipMemcpy(gpuZ, cpuZ, outputSize*sizeof(float), hipMemcpyHostToDevice);
        

    MULBLOCKSX = ceil((float)COORDS/b);
    MULBLOCKSY = ceil((float)OUT_DIM/b);
    dim3 blocks(MULBLOCKSX, MULBLOCKSY);
    MatrixMultiply<<<blocks,threads>>>(COORDS, OUT_DIM, DIM, gpuX, DIM, gpuW, OUT_DIM, gpuZ, OUT_DIM);
    hipDeviceSynchronize();
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);

    idx = 0;
//    for(int i=0;i<COORDS;i++){
//    	for(int j=0;j<OUT_DIM;j++){
//    		cout<<Z[idx++]<<endl;
//    	}
//    }
	cout<<"Time Taken: "<<time/1000<<endl;

  delete [] cpuW;
  delete [] cpuZ;
  delete [] cpuB;
  delete [] cpuX;

  hipFree(gpuW);
  hipFree(gpuZ);
  hipFree(gpuB);
  hipFree(gpuX);
}
