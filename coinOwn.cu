#include <cstdio> 
#include <cstdlib> 
#include <cmath>
#include <cstring>
#include <string>
#include <cassert>
#include <iostream>
#include <fstream>
#include <hip/hip_runtime.h>
#include <hipblas.h>


using namespace std;
__global__ void MatrixMultiply(int M, int N, int K, float* A, int LDA, float* B, int LDB, float*C, int LDC) {
  __shared__ float ABlock[32*32];
  __shared__ float BBlock[32*32];
  int b = 32;
  int tx = threadIdx.x;
  int ty = threadIdx.y;
  int bx = blockIdx.x;
  int by = blockIdx.y;
  int i, j, X_MAX, Y_MAX, K_MAX;
  i = bx*b;
  j = by*b;
  int xoffset = i+tx;
  int yoffset = j+ty;
  int xbase = tx*b;
  int xbaselong = tx*LDB;
  int fixedbaseXA = xoffset*LDA;
  int fixedbaseXC = xoffset*LDC;
  int increment = b*LDB;
  int kbase, fixedbaseKA, fixedbaseKB;
  int cell = xbase + ty;
  X_MAX = min(M, i+b);
  Y_MAX = min(N, j+b);
  int outOfBoundX = xoffset>=X_MAX;
  int outOfBoundY = yoffset>=Y_MAX;
  if(outOfBoundX && outOfBoundY){
    return;
  }
  float temp = 0;
  fixedbaseKA = fixedbaseXA + ty;
  fixedbaseKB = xbaselong+yoffset;
  for(int k=0;k<K;k+=b){
    K_MAX = min(K, k+b)-k;
    if(!outOfBoundX && ty<K_MAX){
      ABlock[cell] = A[fixedbaseKA];
    }
    if(!outOfBoundY && tx<K_MAX){
      BBlock[cell] = B[fixedbaseKB];
    }
    __syncthreads();
    if(!outOfBoundX && !outOfBoundY){
      kbase = ty;
      for(int koffset=0;koffset<K_MAX;koffset++){
        temp += ABlock[xbase+koffset]*BBlock[kbase];
        kbase += b;
      }
    }
    __syncthreads(); 
    fixedbaseKB += increment;
    fixedbaseKA += b;
  }
  if(!outOfBoundX && !outOfBoundY){
    C[fixedbaseXC+yoffset] += temp;
  }
}
__global__ void sineActivation(float *O, float *Z, int N, float weight=30.0) {
    int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
    if(tid<N){
        O[tid] = sin(weight*Z[tid]);
    }
}
void readIntoArray(float* arr, ifstream* inFile, int SIZE){
	if (inFile->is_open())  
    {   

        for (int i = 0; i < SIZE; i++) 
        {
            *inFile >> arr[i];
        }
        inFile->close();
    }
}
void fillCoordinateMatrix(float* X, int STARTX, int STARTY, int ENDX, int ENDY, int RESX, int RESY, int HEIGHT, int WIDTH){
    float start_x = STARTX/(HEIGHT-1.0);
    start_x -= 0.5;
    start_x *= 2.0;
    float start_y = STARTY/(HEIGHT-1.0);
    start_y -= 0.5;
    start_y *= 2.0;
    float diff_x = 2*((ENDX-STARTX)/(HEIGHT-1.0))/RESX;
    float diff_y = 2*((ENDY-STARTY)/(HEIGHT-1.0))/RESY;
    int idx=0;
    float tmp = start_y;
    for(int i=0;i<RESX;i++){
        for(int j=0;j<RESY;j++){
            X[idx++] = start_x;
            X[idx++] = tmp;
            tmp += diff_y;
        }
        start_x += diff_x;
        tmp = start_y;
    }
}
int main(int argc, char* argv[]){

    int INP_DIM = 2;
    int OUT_DIM = 3;

    // ArgParse
    int NUM_LAYERS, DIM, HEIGHT, WIDTH, RESX, RESY, STARTX, STARTY, ENDX, ENDY;
    NUM_LAYERS = atoi(argv[1]);
    DIM = atoi(argv[2]);
    HEIGHT = atoi(argv[3]);
    WIDTH = atoi(argv[4]);
    RESX = atoi(argv[5]);
    RESY = atoi(argv[6]);
    STARTX = atoi(argv[7]);
    STARTY = atoi(argv[8]);
    ENDX = atoi(argv[9]);
    ENDY = atoi(argv[10]);
    
    ifstream inFile;
	  float* W;
  	float* B;
  	float* Z;
  	float* X;
    
    int weightSize = DIM*DIM;
    int biasSize = DIM;
    int COORDS = RESX*RESY;
    int outputSize = COORDS*DIM;
    
    int idx = 0;
    int NUM_THREADS=1024;
    int NUM_BLOCKS;
	  int b=32;
	  int MULTHREADS = 32;
    int MULBLOCKSX;
    int MULBLOCKSY;
    dim3 threads(MULTHREADS, MULTHREADS);
      
    float time;
    hipEvent_t start, stop;	
    
    hipMallocManaged(&Z, outputSize*sizeof(float));
    hipMallocManaged(&W, weightSize*sizeof(float));
    hipMallocManaged(&B, biasSize*sizeof(float));
    hipMallocManaged(&X, COORDS*DIM*sizeof(float));


    fillCoordinateMatrix(X, STARTX, STARTY, ENDX, ENDY, RESX, RESY, HEIGHT, WIDTH);
    
  	hipEventCreate(&start);
  	hipEventCreate(&stop);
  	hipEventRecord(start, 0);

    for(int layer=0;layer<NUM_LAYERS;layer++){
        string weightsfileName = "weightsT/net."+to_string(layer)+".linear.weight";
        string biasfileName = "weightsT/net."+to_string(layer)+".linear.bias";
        inFile.open(weightsfileName.c_str());
        if(layer == 0){
            readIntoArray(W, &inFile, DIM*INP_DIM);
        }
        else{
            readIntoArray(W, &inFile, weightSize);
        }
        inFile.open(biasfileName.c_str());
        readIntoArray(B, &inFile, biasSize);

        idx=0;
        for(int j=0;j<COORDS;j++){
            for(int i=0;i<biasSize;i++){
        		Z[idx++] = B[i];
        	}
        }
        MULBLOCKSX = ceil((float)COORDS/b);
        MULBLOCKSY = ceil((float)DIM/b);
        dim3 blocks(MULBLOCKSX, MULBLOCKSY);
        if(layer == 0){
            MatrixMultiply<<<blocks,threads>>>(COORDS, DIM, INP_DIM, X, INP_DIM, W, DIM, Z, DIM);
        }
        else{
            MatrixMultiply<<<blocks,threads>>>(COORDS, DIM, DIM, X, DIM, W, DIM, Z, DIM);
        }
        hipDeviceSynchronize();
        NUM_BLOCKS=ceil((float)(COORDS*DIM)/NUM_THREADS);
        sineActivation<<<NUM_BLOCKS, NUM_THREADS>>>(X, Z, COORDS*DIM);
        hipDeviceSynchronize();
    }
    string weightsfileName = "weightsT/last_layer.linear.weight";
    string biasfileName = "weightsT/last_layer.linear.bias";
    inFile.open(weightsfileName.c_str());
    readIntoArray(W, &inFile, DIM*OUT_DIM);
    inFile.open(biasfileName.c_str());
    readIntoArray(B, &inFile, OUT_DIM);
    idx=0;

    for(int j=0;j<COORDS;j++){
        for(int i=0;i<biasSize;i++){
            Z[idx++] = B[i];
        }
    }
    MULBLOCKSX = ceil((float)COORDS/b);
    MULBLOCKSY = ceil((float)OUT_DIM/b);
    dim3 blocks(MULBLOCKSX, MULBLOCKSY);
    MatrixMultiply<<<blocks,threads>>>(COORDS, OUT_DIM, DIM, X, DIM, W, OUT_DIM, Z, OUT_DIM);
    hipDeviceSynchronize();
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);

    idx = 0;
//    for(int i=0;i<COORDS;i++){
//    	for(int j=0;j<OUT_DIM;j++){
//    		cout<<Z[idx++]<<endl;
//    	}
//    }
	cout<<"Time Taken: "<<time/1000<<endl;

    hipFree(W);
    hipFree(Z);
    hipFree(B);
    hipFree(X);
}
