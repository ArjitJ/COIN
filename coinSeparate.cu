#include <cstdio> 
#include <cstdlib> 
#include <cmath>
#include <cstring>
#include <string>
#include <cassert>
#include <iostream>
#include <fstream>
#include <hip/hip_runtime.h>
#include <hipblas.h>


using namespace std;
__global__ void copyBias(float *O, float *Z, int N, int M) {
    int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
    if(tid<N){
        O[tid] = Z[tid%M];
    }
}
__global__ void sineActivation(float *O, float *Z, int N, float weight=30.0) {
    int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
    if(tid<N){
        O[tid] = sin(weight*Z[tid]);
    }
}
void readIntoArray(float* arr, ifstream* inFile, int SIZE){
    if (inFile->is_open())  
    {
        for (int i = 0; i < SIZE; i++) 
        {
            *inFile >> arr[i];
        }
        inFile->close();
    }
}
__global__ void fillCoordinateMatrixCUDA(float* X, float start_x, float start_y, float diff_x, float diff_y, int RESX, int RESY){
    int idx;
    int tidx = (blockIdx.x * blockDim.x) + threadIdx.x;
    int tidy = (blockIdx.y * blockDim.y) + threadIdx.y;
    if(tidx < RESX && tidy < RESY){
        idx = 2*(tidx*RESY + tidy);
        X[idx++] = start_x + tidx*(diff_x);
        X[idx++] = start_y + tidy*(diff_y);
    }
}
void fillCoordinateMatrix(float* X, int STARTX, int STARTY, int ENDX, int ENDY, int RESX, int RESY, int HEIGHT, int WIDTH){
    float start_x = STARTX/(HEIGHT-1.0);
    start_x -= 0.5;
    start_x *= 2.0;
    float start_y = STARTY/(HEIGHT-1.0);
    start_y -= 0.5;
    start_y *= 2.0;
    float diff_x = 2*((ENDX-STARTX)/(HEIGHT-1.0))/RESX;
    float diff_y = 2*((ENDY-STARTY)/(HEIGHT-1.0))/RESY;
    int idx=0;
    float tmp = start_y;
    for(int i=0;i<RESX;i++){
        for(int j=0;j<RESY;j++){
            X[idx++] = start_x;
            X[idx++] = tmp;
            tmp += diff_y;
        }
        start_x += diff_x;
        tmp = start_y;
    }
}
int main(int argc, char* argv[]){

    int INP_DIM = 2;
    int OUT_DIM = 3;

    // ArgParse
int NUM_LAYERS, DIM, HEIGHT, RESX, RESY, STARTX, STARTY, ENDX, ENDY, PRINT_TIME;
    NUM_LAYERS = atoi(argv[1]);
    DIM = atoi(argv[2]);
    HEIGHT = atoi(argv[3]);
    RESX = atoi(argv[4]);
    RESY = atoi(argv[5]);
    STARTX = atoi(argv[6]);
    STARTY = atoi(argv[7]);
    ENDX = atoi(argv[8]);
    ENDY = atoi(argv[9]);
    PRINT_TIME = atoi(argv[10]);
    
    ifstream inFile;
    float* cpuW;
    float* cpuB;
    float* cpuZ;
    float* cpuX;
    float* gpuW;
    float* gpuB;
    float* gpuZ;
    float* gpuX;
    int weightSize = DIM*DIM;
    int biasSize = DIM;
    int COORDS = RESX*RESY;
    int outputSize = COORDS*DIM;
    float alpha = 1.0f;
    float beta = 1.0f;
    
    int idx = 0;
    int NUM_THREADS=1024;
    int NUM_BLOCKS;
    
    float start_x = STARTX/(HEIGHT-1.0);
    start_x -= 0.5;
    start_x *= 2.0;
    float start_y = STARTY/(HEIGHT-1.0);
    start_y -= 0.5;
    start_y *= 2.0;
    float diff_x = 2*((ENDX-STARTX)/(HEIGHT-1.0))/RESX;
    float diff_y = 2*((ENDY-STARTY)/(HEIGHT-1.0))/RESY;
    float time;
    hipEvent_t start, stop;    
    
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    cpuZ = new float[outputSize];
    cpuW = new float[weightSize];
    cpuB = new float[biasSize];
    cpuX = new float[COORDS*DIM];
    
    hipMalloc(&gpuZ, outputSize*sizeof(float));
    hipMalloc(&gpuW, weightSize*sizeof(float));
    hipMalloc(&gpuB, biasSize*sizeof(float));
    hipMalloc(&gpuX, COORDS*DIM*sizeof(float));

    dim3 threads(32, 32);
    dim3 blocks(ceil((float)RESX/32), ceil((float)RESY/32));
    fillCoordinateMatrixCUDA<<<blocks, threads>>>(gpuX, start_x, start_y, diff_x, diff_y, RESX, RESY);
    hipDeviceSynchronize();
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    for(int layer=0;layer<NUM_LAYERS;layer++){
    
        string weightsfileName = "weightsT/net."+to_string(layer)+".linear.weight";
        string biasfileName = "weightsT/net."+to_string(layer)+".linear.bias";
        inFile.open(weightsfileName.c_str());
        if(layer == 0){
            readIntoArray(cpuW, &inFile, DIM*INP_DIM);
        }
        else{
            readIntoArray(cpuW, &inFile, weightSize);
        }
        inFile.open(biasfileName.c_str());
        readIntoArray(cpuB, &inFile, biasSize);
        hipMemcpy(gpuW, cpuW, weightSize*sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(gpuB, cpuB, biasSize*sizeof(float), hipMemcpyHostToDevice);
        copyBias<<<NUM_BLOCKS, NUM_THREADS>>>(gpuZ, gpuB, COORDS*biasSize, biasSize);
        hipDeviceSynchronize();
        if(layer == 0){
            hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, DIM, COORDS, INP_DIM, &alpha, gpuW, DIM, gpuX, INP_DIM,
                    &beta, gpuZ, DIM);
        }
        else{
            hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, DIM, COORDS, DIM, &alpha, gpuW, DIM, gpuX, DIM,
                    &beta, gpuZ, DIM);
        }

        hipDeviceSynchronize();
        NUM_BLOCKS=ceil((float)(COORDS*DIM)/NUM_THREADS);
        sineActivation<<<NUM_BLOCKS, NUM_THREADS>>>(gpuX, gpuZ, COORDS*DIM);
        hipDeviceSynchronize();
    }

    string weightsfileName = "weightsT/last_layer.linear.weight";
    string biasfileName = "weightsT/last_layer.linear.bias";
    inFile.open(weightsfileName.c_str());
    readIntoArray(cpuW, &inFile, DIM*OUT_DIM);
    inFile.open(biasfileName.c_str());
    readIntoArray(cpuB, &inFile, OUT_DIM);
    idx=0;
    hipMemcpy(gpuW, cpuW, weightSize*sizeof(float), hipMemcpyHostToDevice);
    
    hipMemcpy(gpuB, cpuB, biasSize*sizeof(float), hipMemcpyHostToDevice);
    copyBias<<<NUM_BLOCKS, NUM_THREADS>>>(gpuZ, gpuB, COORDS*biasSize, biasSize);
    hipDeviceSynchronize();

        
    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, OUT_DIM, COORDS, DIM, &alpha, gpuW, OUT_DIM, gpuX, DIM,
            &beta, gpuZ, OUT_DIM);
    hipDeviceSynchronize();

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    
    hipMemcpy(cpuZ, gpuZ, outputSize*sizeof(float), hipMemcpyDeviceToHost);
    if(PRINT_TIME){
        cout<<"Time Taken: "<<time/1000<<endl;
    }
    else{
        idx = 0;
        for(int i=0;i<COORDS;i++){
            for(int j=0;j<OUT_DIM;j++){
                cout<<cpuZ[idx++]<<endl;
            }
        }
    }

    delete [] cpuW;
    delete [] cpuZ;
    delete [] cpuB;
    delete [] cpuX;
    
    hipFree(gpuW);
    hipFree(gpuZ);
    hipFree(gpuB);
    hipFree(gpuX);
}
